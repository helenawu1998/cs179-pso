#include "hip/hip_runtime.h"
/*
 * CUDA Particle Swarm Optimization
 * Helena Wu
 * CS 179 Final Project
 */

#include "pso.cuh"
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <cstring>
#include <iostream>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
// #include "benchmark_functions.h"
#include "cuda_header.cuh"
#define PI 3.14159265358979

/* Benchmark optimization problem known in literature as Rosenbrock's function.
 * The minimum of the function is at (1, 1) with value 0. We search in the
 * domain [-5.12, 5.12]^2 as in other benchmark tests.
 */
CUDA_CALLABLE float rosenbrock(float* solution) {
    // Penalize solutions that are out of the domain
    if (solution[0] < -5.12 || solution[0] > 5.12 || solution[1] < -5.12 || solution[1] > 5.12 ){
        return 1000;
    }
    return (100 * pow(pow(solution[0], 2) - solution[1], 2) +
        pow(1 - solution[0], 2));
}

/* Benchmark optimization problem known in literature as the Rastrigin function.
 * The minimum of the function is at (0, 0, 0 ..) with value 0. We search in the
 * domain [-5.12, 5.12]^dim as in other benchmark tests.
 */
CUDA_CALLABLE float rastrigin(float* solution, int dim) {
    float ans = 10 * dim;
    // Penalize solutions that are out of the domain. Otherwise compute objective.
    for (int i = 0; i < dim; i++) {
        if (solution[i] < -5.12 || solution[i] > 5.12)
            return 1000;
        ans += pow(solution[i], 2) - 10 * cos(2 * PI * solution[i]);
    }
    return ans;
}

/* Returns the value of the objective function, which we are trying to minimize.
 * User defines which objective function to use for benchmark tests.
 */
CUDA_CALLABLE float cost(int objective, float* solution, int dim) {
    if (objective == 0)
        return rosenbrock(solution);
    else {
        return rastrigin(solution, dim);
    }
}

/* Returns 1 if solution 1 < solution 2, and returns 0 otherwise.
 */
CUDA_CALLABLE float is_min_cost(int objective, float* solution1, float* solution2, int dim) {
    if (cost(objective, solution1, dim) < cost(objective, solution2, dim)) {
        return 1;
    }
    return 0;
}

/* This function will run the PSO algorithm until the convergence condition is
 * reached. Each thread represents one particle of the swarm and will do all
 * computations for its position and velocity updates.
 */
__global__
void cuda_pso_kernel(float *gpu_solutions, float *gpu_velocities,
                      float *gpu_p_best, float *gpu_g_best,
                      const int num_particles, const int dim,
                      const int benchmark, const float c1,
                      const float c2, const float w) {

    extern __shared__ float sdata[];
    //       Compute the current thread index.
    uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    uint tid = threadIdx.x;

    //       Init for random number generation
    hiprandState state;
    hiprand_init(clock64(), thread_index, 0, &state);

    //  First initialize the personal bests with input solutions
    gpu_p_best[thread_index] = gpu_solutions[thread_index];

    float r1;
    float r2;
    float temp;

    //  Repeat until stopping criteria is satisfied
    while (abs(cost(benchmark, gpu_g_best, dim)) > 0.0001) {

        // r1, r2 are random floats between (0,1] for velocity update equation
        r1 = hiprand_uniform(&state);
        r2 = hiprand_uniform(&state);
        // Latency hiding with arithmetic operations and memory accesses
        for (int dim_idx = 0; dim_idx < dim; dim_idx++) {
            uint i = thread_index * dim + dim_idx;

            // Update particle's velocity
            temp = 0;
            temp += w * gpu_velocities[i];
            temp += c1 * r1 * (gpu_p_best[i] - gpu_solutions[i]);
            temp += c2 * r2 * (gpu_g_best[i] - gpu_solutions[i]);
            gpu_velocities[i] = temp;

            // Update particle's position
            gpu_solutions[i] += temp;
            // Store solutions for this block in shared memory
            sdata[tid * dim + dim_idx] = gpu_solutions[i];
        }
        // Compute cost of updated solution
        float solution_cost = cost(benchmark, &gpu_solutions[thread_index], dim);
        // Update personal best if better
        if (solution_cost < cost(benchmark, &gpu_p_best[thread_index], dim)) {
            for (int i = 0; i < dim; i++) {
                gpu_p_best[thread_index + i] = gpu_solutions[thread_index + i];
            }
        }

        // Synchronize so all threads in block have completed the iteration and
        // stored it in shared memory
        __syncthreads();

        // Each thread does reduction in shared mem
        // Reference: "Optimizing Parallel Reduction in CUDA" by Mark Harris
        for (uint s = blockDim.x/2; s > 0; s >>= 1){
            if (tid < s){
                if (is_min_cost(benchmark, &sdata[tid*dim + s*dim], &sdata[tid*dim], dim)) {
                    // Copy solution elements to smaller index for reduction
                    for (int i = 0; i < dim; i++) {
                        sdata[tid*dim +i] = sdata[tid*dim + s*dim + i];
                    }
                }
            }
            // Synchronize threads between reduction steps
            __syncthreads();
        }

        // Write result to gpu_g_best if minimum
        if (tid == 0 && is_min_cost(benchmark, &sdata[0], gpu_g_best, dim)) {
            for (int i = 0; i < dim; i++) {
                gpu_g_best[i] = sdata[i];
            }
        }
    }
}

/* This function will be in charge of allocating GPU memory, invoking the kernel
 * for running the PSO algorithm, and cleaning up afterwards. The result
 * will be stored in out_data. The function returns the amount of time that
 * it took for the function to complete (prior to returning) in milliseconds.
 */
float cuda_call_pso_kernel(const unsigned int blocks,
                           const unsigned int threads_per_block,
                           float *in_solutions, float *in_velocities,
                           float *out_data, const unsigned int num_particles,
                           const unsigned int dim, const int benchmark,
                           const float c1, const float c2, const float w) {

    //       Use the CUDA machinery for recording time
    hipEvent_t start_gpu, stop_gpu;
    float time_milli = -1;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);
    hipEventRecord(start_gpu);

    //       Allocate GPU memory for the raw input data (randomly generated
    //       solutions, velocities in the initial population).
    //       The data is of type float and has dim * num_particles elements.
    float* gpu_solutions;
    gpu_errchk(hipMalloc((void **) &gpu_solutions,
        dim * num_particles * sizeof(float)));
    gpu_errchk(hipMemcpy(gpu_solutions, in_solutions,
        dim * num_particles * sizeof(float), hipMemcpyHostToDevice));

    float* gpu_velocities;
    gpu_errchk(hipMalloc((void **) &gpu_velocities,
        dim * num_particles * sizeof(float)));
    gpu_errchk(hipMemcpy(gpu_velocities, in_velocities,
        dim * num_particles * sizeof(float), hipMemcpyHostToDevice));

    //       Allocate GPU memory to store the personal best solution for each
    //       particle in the swarm.
    float* gpu_p_best;
    gpu_errchk(hipMalloc((void **) &gpu_p_best,
        dim * num_particles * sizeof(float)));

    //       Allocate GPU memory to store the global best solution, with dim
    //       number of elements of type float.
    float* gpu_g_best;
    gpu_errchk(hipMalloc((void **) &gpu_g_best, dim * sizeof(float)));
    //      Initialize the gpu_g_best with first solution in population, as
    //      currently stored in out_data.
    gpu_errchk(hipMemcpy(gpu_g_best, out_data,
        dim * sizeof(float), hipMemcpyHostToDevice));

    //      Call the kernel function.
    cuda_pso_kernel<<<blocks, threads_per_block, threads_per_block * dim * sizeof(float)>>>
        (gpu_solutions, gpu_velocities, gpu_p_best, gpu_g_best, num_particles, dim,
        benchmark, c1, c2, w);

    //      Check for errors on kernel call
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
    else
        fprintf(stderr, "No kernel error detected\n");

    //       Now that kernel calls have finished, copy the output signal
    //       back from the GPU to host memory.
    gpu_errchk(hipMemcpy(out_data, gpu_g_best, dim * sizeof(float),
        hipMemcpyDeviceToHost));

    //       Now that we have finished our computations on the GPU, free the
    //       GPU resources.
    gpu_errchk(hipFree(gpu_solutions));
    gpu_errchk(hipFree(gpu_velocities));
    gpu_errchk(hipFree(gpu_g_best));
    gpu_errchk(hipFree(gpu_p_best));

    //      Stop the recording timer and return the computation time
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
    hipEventElapsedTime(&time_milli, start_gpu, stop_gpu);
    return time_milli;
}
